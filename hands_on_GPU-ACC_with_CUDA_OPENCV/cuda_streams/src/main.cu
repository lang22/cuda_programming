// CUDA streams for vector addition
// Author: Yuting Xie
// 2022.2.22


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (15000)

__global__ void vector_add(int *v1, int *v2, int *res) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		res[tid] = v1[tid] + v2[tid];
	}
}

int main(void) {
	int v1[2 * N], v2[2 * N], res[2 * N];
	for (int i = 0; i < 2 * N; ++i) {
		v1[i] = v2[i] = i;
	}

	int *d_v1, *d_v2, *d_res;
	hipMalloc((void**)&d_v1, 2 * N * sizeof(int));
	hipMalloc((void**)&d_v2, 2 * N * sizeof(int));
	hipMalloc((void**)&d_res, 2 * N * sizeof(int));

	// Create CUDA streams
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	// Copy memory with streams use async version
	hipMemcpyAsync(d_v1, v1, N * sizeof(int), hipMemcpyHostToDevice, stream0); // This async operation assigned to stream0
	hipMemcpyAsync(d_v2, v2, N * sizeof(int), hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(d_v1 + N, v1 + N, N * sizeof(int), hipMemcpyHostToDevice, stream1); // This async operation assigned to stream1
	hipMemcpyAsync(d_v2 + N, v2 + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

	// Complete kernel call form <<<Block, Threads, Shm_size, Stream>>>
	vector_add<<<(N + 127) / 128, 128, 0, stream0>>>(d_v1, d_v2, d_res);
	vector_add<<<(N + 127) / 128, 128, 0, stream1>>>(d_v1 + N, d_v2 + N, d_res + N);

	// Collect results
	hipMemcpyAsync(res, d_res, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
	hipMemcpyAsync(res + N, d_res + N, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

	// Wait all streams to finish their CE and KE jobs
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	for (int i = 0; i < 20; ++i) {
		printf("%d, ", res[i]);
	}

	return 0;
}
