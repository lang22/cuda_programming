// Timing kernel executions with cudaEvent
// Author: Yuting Xie
// 2022.2.21


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <algorithm>

#define N (1 << 20)

__global__
void saxpy(float a, float *x, float *y) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		y[idx] = a * x[idx] + y[idx];
	}
}

int main(void) {
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));
	for (int i = 0; i < N; ++i) {
		x[i] = 1.f;
		y[i] = 2.f;
	}

	hipMalloc((void**)&d_x, N * sizeof(float));
	hipMalloc((void**)&d_y, N * sizeof(float));

	// Create cudaEvent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

	// The following 3 lines are to be put into NULL stream, async.
	// The device will record a time stamp for the event when it reaches that event in the stream
	hipEventRecord(start);
	saxpy<<<(N + 1023) / 1024, 1024>>>(2.f, d_x, d_y);
	hipEventRecord(stop);

	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	// Important! Since the cudaEventRecord is async! Have to wait till "stop" happens
	hipEventSynchronize(stop);

	// Examine the timing result
	float msecs;
	hipEventElapsedTime(&msecs, start, stop);
	printf("Kernel executes %.3f ms\n", msecs);

	float maxErr = 0.f;
	for (int i = 0; i < N; ++i) {
		maxErr = std::max(maxErr, std::fabs(4.f - y[i]));
	}
	printf("Max error is %.6f\n", maxErr);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}
